#include "hip/hip_runtime.h"
#include "internal.hpp"
using namespace cv;
using namespace cv::cuda;

const int THREADS_H = 16;
const int THREADS_W = 32;
const int WINDOW_H = 5;
const int WINDOW_W = 7;
const int P1 = 25;
const int P2 = 100;
typedef short cost_type;
__device__ cost_type lr_array[8*WIDTH*HEIGHT*D_MAX];


__global__ void diff_census_kernel(const uint* left_census,const uint*  right_census,uchar *cost_array);



namespace mycuda{
    __device__ short min(short x1,short x2){
        if(x1>=x2){
            return x2;
        }
        return x1;
    }
    __device__ short max(short x1,short x2){
        if(x1<=x2){
            return x2;
        }
        return x1;
    }
}

/* kernels */
//display_range threads per block * grid( nx * ny )
__global__ void diff_kernel(const uchar* left,const uchar* __restrict__ right,uchar *cost_array)
{
	__shared__ uchar right_s[HEIGHT_SINGLE][2*WIDTH+2];
	uchar value_left[HEIGHT_SINGLE];
    const int y_base = (blockIdx.y*blockDim.y+threadIdx.y)*HEIGHT_SINGLE;
	const int x = (blockIdx.x*blockDim.x+threadIdx.x)+D_MAX;
	const int x_s = x<<1;
	if (y_base < HEIGHT){
		#pragma unroll
		for(int y=0;y<HEIGHT_SINGLE;y++){
			const int pos_left = (y_base+y) * WIDTH + x;
			value_left[y] = left[pos_left];
			right_s[y][x_s+1] = right[pos_left];
			if(x == WIDTH-1){
				right_s[y][x_s+2] = right[pos_left];
			}else{
				right_s[y][x_s+2] = (right[pos_left]+right[pos_left+1])/2;
			}
			if(x < 2 * D_MAX){
				right_s[y][((x - D_MAX)<<1)+1] = right[pos_left - D_MAX];
				if(x == D_MAX){
					right_s[y][0] = right[pos_left - D_MAX];
				}
				right_s[y][((x - D_MAX)<<1)+2] = (right[pos_left - D_MAX]+right[pos_left - D_MAX+1])/2;
			}
		}
		__syncthreads();
		int right_pos,pos_l,pos_m,pos_r,right_max,right_min;
		#pragma unroll
		for(int disparity=0;disparity<D_MAX;disparity++){
		    for(int y=0;y<HEIGHT_SINGLE;y++){			
			    const int pos_left = (y_base+y) * WIDTH + x;
			    const int value_left_t = value_left[y];
			    if(x < WIDTH && y_base+y < HEIGHT) {
				    //#pragma unroll
					right_pos = ((x-disparity)<<1);
					pos_l = right_s[y][right_pos-1];
					pos_m = right_s[y][right_pos];
					pos_r = right_s[y][right_pos+1];
	
					right_min = min(pos_m,pos_l);
					right_min = min(right_min,pos_r);
					right_max = max(pos_m,pos_l);
					right_max = max(right_max,pos_r);

				    int diff = max(0,value_left_t-right_max);
				    diff = max(diff,right_min-value_left_t);
				    cost_array[pos_left+FRAME_SIZE*disparity] = diff;
				}
			}
		}
	}
}

//测试 单一路径规划
__global__ void dp_1_kernel(short* dp_array,uchar* cost_all){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    if(x < WIDTH){
        //y = 0
        {
            //d = 0
            short cost = cost_all[x];
            last_line_disparity_last[0] = cost;
            dp_array[x] = cost;
            last_best_disparity_pos = 0;
            last_best_disparity_value = cost;
            //d > 0
            #pragma unroll
            for(int d=1;d<D_MAX;d++){
                //当前代价
                short cost = cost_all[x+d*FRAME_SIZE];
                last_line_disparity_last[d] = cost;
                dp_array[x+FRAME_SIZE*d] = cost;
                //选取最好的保存
                if(cost < last_best_disparity_value){
                    last_best_disparity_pos = d;
                    last_best_disparity_value = cost;
                }
            }
        }
        //y > 0
        #pragma unroll
        for(int y=1;y<HEIGHT;y++){
            const int pos = y * WIDTH + x;
            //d = 0
            {
                short cost = cost_all[pos];
                //no left disparity
                ///int left_t = P1+last_line_disparity[d-1];
                //mid
                short mid_t = last_line_disparity_last[0];
                //right
                short right_t = last_line_disparity_last[0+1]+P1;
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
                short d0_value = cost+min_t-last_best_disparity_value;
                dp_array[pos] = d0_value;
                now_best_disparity_pos = 0;
                now_best_disparity_value = d0_value;
                last_line_disparity_now[0] = d0_value;
            }
            #pragma unroll
            for(int d=1;d<D_MAX-1;d++){
                int pos_d = pos+FRAME_SIZE*d;
                short cost = cost_all[pos_d];
                short left_t = last_line_disparity_last[d-1]+P1;
                short mid_t = last_line_disparity_last[d];
                short right_t = last_line_disparity_last[d+1]+P1;
                short last_t = last_best_disparity_value+P2;
                //get min of t
                int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
                short di_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = di_value;
                last_line_disparity_now[d] = di_value;
                if(di_value < now_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = di_value;
                }
            }
            //d = D_MAX-1
            {
                int d = D_MAX-1;
                int pos_d = pos+FRAME_SIZE*(D_MAX-1);
                short cost = cost_all[pos_d];
                //no left disparity
                short left_t = last_line_disparity_last[d-1]+P1;
                //mid
                short mid_t = last_line_disparity_last[d];
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
                short de_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = de_value;
                last_line_disparity_now[d] = de_value;
                if(de_value < last_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = de_value;
                }
            }
            //swap
            {
                short *p_t = last_line_disparity_now;
                last_line_disparity_now = last_line_disparity_last;
                last_line_disparity_last = p_t;
                last_best_disparity_pos = now_best_disparity_pos;
                last_best_disparity_value = now_best_disparity_value;
            }
        }
    }
}

//测试 单一路径规划
__global__ void dp_2_kernel(short* dp_array,uchar* cost_all){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    if(x < WIDTH){
        //y = 0
        {
            //d = 0
            int pos = x+(D_MAX-1)*WIDTH;
            short cost = cost_all[pos];
            last_line_disparity_last[0] = cost;
            dp_array[pos] = cost;
            last_best_disparity_pos = 0;
            last_best_disparity_value = cost;
            //d > 0
            #pragma unroll
            for(int d=1;d<D_MAX;d++){
                //当前代价
                short cost = cost_all[pos];
                last_line_disparity_last[d] = cost;
                dp_array[pos+FRAME_SIZE*d] = cost;
                //选取最好的保存
                if(cost < last_best_disparity_value){
                    last_best_disparity_pos = d;
                    last_best_disparity_value = cost;
                }
            }
        }
        //y > 0
        #pragma unroll
        for(int y=HEIGHT-2;y>=0;y--){
            const int pos = y * WIDTH + x;
            //d = 0
            {
                short cost = cost_all[pos];
                //no left disparity
                ///int left_t = P1+last_line_disparity[d-1];
                //mid
                short mid_t = last_line_disparity_last[0];
                //right
                short right_t = last_line_disparity_last[0+1]+P1;
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
                short d0_value = cost+min_t-last_best_disparity_value;
                dp_array[pos] = d0_value;
                now_best_disparity_pos = 0;
                now_best_disparity_value = d0_value;
                last_line_disparity_now[0] = d0_value;
            }
            #pragma unroll
            for(int d=1;d<D_MAX-1;d++){
                int pos_d = pos+FRAME_SIZE*d;
                short cost = cost_all[pos_d];
                short left_t = last_line_disparity_last[d-1]+P1;
                short mid_t = last_line_disparity_last[d];
                short right_t = last_line_disparity_last[d+1]+P1;
                short last_t = last_best_disparity_value+P2;
                //get min of t
                int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
                short di_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = di_value;
                last_line_disparity_now[d] = di_value;
                if(di_value < now_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = di_value;
                }
            }
            //d = D_MAX-1
            {
                int d = D_MAX-1;
                int pos_d = pos+FRAME_SIZE*(D_MAX-1);
                short cost = cost_all[pos_d];
                //no left disparity
                short left_t = last_line_disparity_last[d-1]+P1;
                //mid
                short mid_t = last_line_disparity_last[d];
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
                short de_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = de_value;
                last_line_disparity_now[d] = de_value;
                if(de_value < last_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = de_value;
                }
            }
            //swap
            {
                short *p_t = last_line_disparity_now;
                last_line_disparity_now = last_line_disparity_last;
                last_line_disparity_last = p_t;
                last_best_disparity_pos = now_best_disparity_pos;
                last_best_disparity_value = now_best_disparity_value;
            }
        }
    }
}
//测试 单一路径规划
__global__ void dp_3_kernel(short* dp_array,uchar* cost_all){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    if(x < WIDTH){
        //y = 0
        {
            //d = 0
            int pos = x+(D_MAX-1)*WIDTH;
            short cost = cost_all[pos];
            last_line_disparity_last[0] = cost;
            dp_array[pos] = cost;
            last_best_disparity_pos = 0;
            last_best_disparity_value = cost;
            //d > 0
            #pragma unroll
            for(int d=1;d<D_MAX;d++){
                //当前代价
                short cost = cost_all[pos];
                last_line_disparity_last[d] = cost;
                dp_array[pos+FRAME_SIZE*d] = cost;
                //选取最好的保存
                if(cost < last_best_disparity_value){
                    last_best_disparity_pos = d;
                    last_best_disparity_value = cost;
                }
            }
        }
        //y > 0
        #pragma unroll
        for(int y=HEIGHT-2;y>=0;y--){
            x--;
            if(x<0){
                return;
            }
            const int pos = y * WIDTH + x;
            //d = 0
            {
                short cost = cost_all[pos];
                //no left disparity
                ///int left_t = P1+last_line_disparity[d-1];
                //mid
                short mid_t = last_line_disparity_last[0];
                //right
                short right_t = last_line_disparity_last[0+1]+P1;
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
                short d0_value = cost+min_t-last_best_disparity_value;
                dp_array[pos] = d0_value;
                now_best_disparity_pos = 0;
                now_best_disparity_value = d0_value;
                last_line_disparity_now[0] = d0_value;
            }
            #pragma unroll
            for(int d=1;d<D_MAX-1;d++){
                int pos_d = pos+FRAME_SIZE*d;
                short cost = cost_all[pos_d];
                short left_t = last_line_disparity_last[d-1]+P1;
                short mid_t = last_line_disparity_last[d];
                short right_t = last_line_disparity_last[d+1]+P1;
                short last_t = last_best_disparity_value+P2;
                //get min of t
                int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
                short di_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = di_value;
                last_line_disparity_now[d] = di_value;
                if(di_value < now_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = di_value;
                }
            }
            //d = D_MAX-1
            {
                int d = D_MAX-1;
                int pos_d = pos+FRAME_SIZE*(D_MAX-1);
                short cost = cost_all[pos_d];
                //no left disparity
                short left_t = last_line_disparity_last[d-1]+P1;
                //mid
                short mid_t = last_line_disparity_last[d];
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
                short de_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = de_value;
                last_line_disparity_now[d] = de_value;
                if(de_value < last_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = de_value;
                }
            }
            //swap
            {
                short *p_t = last_line_disparity_now;
                last_line_disparity_now = last_line_disparity_last;
                last_line_disparity_last = p_t;
                last_best_disparity_pos = now_best_disparity_pos;
                last_best_disparity_value = now_best_disparity_value;
            }
        }
    }
}
__global__ void dp_4_kernel(short* dp_array,uchar* cost_all){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    if(x < WIDTH){
        //y = 0
        {
            //d = 0
            short cost = cost_all[x];
            last_line_disparity_last[0] = cost;
            dp_array[x] = cost;
            last_best_disparity_pos = 0;
            last_best_disparity_value = cost;
            //d > 0
            #pragma unroll
            for(int d=1;d<D_MAX;d++){
                //当前代价
                short cost = cost_all[x+d*FRAME_SIZE];
                last_line_disparity_last[d] = cost;
                dp_array[x+FRAME_SIZE*d] = cost;
                //选取最好的保存
                if(cost < last_best_disparity_value){
                    last_best_disparity_pos = d;
                    last_best_disparity_value = cost;
                }
            }
        }
        //y > 0
        #pragma unroll
        for(int y=1;y<HEIGHT;y++){
            x++;
            if(x>=WIDTH){
                return;
            }
            const int pos = y * WIDTH + x;
            //d = 0
            {
                short cost = cost_all[pos];
                //no left disparity
                ///int left_t = P1+last_line_disparity[d-1];
                //mid
                short mid_t = last_line_disparity_last[0];
                //right
                short right_t = last_line_disparity_last[0+1]+P1;
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
                short d0_value = cost+min_t-last_best_disparity_value;
                dp_array[pos] = d0_value;
                now_best_disparity_pos = 0;
                now_best_disparity_value = d0_value;
                last_line_disparity_now[0] = d0_value;
            }
            #pragma unroll
            for(int d=1;d<D_MAX-1;d++){
                int pos_d = pos+FRAME_SIZE*d;
                short cost = cost_all[pos_d];
                short left_t = last_line_disparity_last[d-1]+P1;
                short mid_t = last_line_disparity_last[d];
                short right_t = last_line_disparity_last[d+1]+P1;
                short last_t = last_best_disparity_value+P2;
                //get min of t
                int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
                short di_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = di_value;
                last_line_disparity_now[d] = di_value;
                if(di_value < now_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = di_value;
                }
            }
            //d = D_MAX-1
            {
                int d = D_MAX-1;
                int pos_d = pos+FRAME_SIZE*(D_MAX-1);
                short cost = cost_all[pos_d];
                //no left disparity
                short left_t = last_line_disparity_last[d-1]+P1;
                //mid
                short mid_t = last_line_disparity_last[d];
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
                short de_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = de_value;
                last_line_disparity_now[d] = de_value;
                if(de_value < last_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = de_value;
                }
            }
            //swap
            {
                short *p_t = last_line_disparity_now;
                last_line_disparity_now = last_line_disparity_last;
                last_line_disparity_last = p_t;
                last_best_disparity_pos = now_best_disparity_pos;
                last_best_disparity_value = now_best_disparity_value;
            }
        }
    }
}
__global__ void dp_5_kernel(short* dp_array,uchar* cost_all){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    if(x < WIDTH){
        //y = 0
        {
            //d = 0
            short cost = cost_all[x];
            last_line_disparity_last[0] = cost;
            dp_array[x] = cost;
            last_best_disparity_pos = 0;
            last_best_disparity_value = cost;
            //d > 0
            #pragma unroll
            for(int d=1;d<D_MAX;d++){
                //当前代价
                short cost = cost_all[x+d*FRAME_SIZE];
                last_line_disparity_last[d] = cost;
                dp_array[x+FRAME_SIZE*d] = cost;
                //选取最好的保存
                if(cost < last_best_disparity_value){
                    last_best_disparity_pos = d;
                    last_best_disparity_value = cost;
                }
            }
        }
        //y > 0
        #pragma unroll
        for(int y=1;y<HEIGHT;y++){
            x--;
            if(x<0){
                return;
            }
            const int pos = y * WIDTH + x;
            //d = 0
            {
                short cost = cost_all[pos];
                //no left disparity
                ///int left_t = P1+last_line_disparity[d-1];
                //mid
                short mid_t = last_line_disparity_last[0];
                //right
                short right_t = last_line_disparity_last[0+1]+P1;
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
                short d0_value = cost+min_t-last_best_disparity_value;
                dp_array[pos] = d0_value;
                now_best_disparity_pos = 0;
                now_best_disparity_value = d0_value;
                last_line_disparity_now[0] = d0_value;
            }
            #pragma unroll
            for(int d=1;d<D_MAX-1;d++){
                int pos_d = pos+FRAME_SIZE*d;
                short cost = cost_all[pos_d];
                short left_t = last_line_disparity_last[d-1]+P1;
                short mid_t = last_line_disparity_last[d];
                short right_t = last_line_disparity_last[d+1]+P1;
                short last_t = last_best_disparity_value+P2;
                //get min of t
                int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
                short di_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = di_value;
                last_line_disparity_now[d] = di_value;
                if(di_value < now_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = di_value;
                }
            }
            //d = D_MAX-1
            {
                int d = D_MAX-1;
                int pos_d = pos+FRAME_SIZE*(D_MAX-1);
                short cost = cost_all[pos_d];
                //no left disparity
                short left_t = last_line_disparity_last[d-1]+P1;
                //mid
                short mid_t = last_line_disparity_last[d];
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
                short de_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = de_value;
                last_line_disparity_now[d] = de_value;
                if(de_value < last_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = de_value;
                }
            }
            //swap
            {
                short *p_t = last_line_disparity_now;
                last_line_disparity_now = last_line_disparity_last;
                last_line_disparity_last = p_t;
                last_best_disparity_pos = now_best_disparity_pos;
                last_best_disparity_value = now_best_disparity_value;
            }
        }
    }
}
__global__ void dp_6_kernel(short* dp_array,uchar* cost_all){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    if(x < WIDTH){
        //y = 0
        {
            //d = 0
            int pos = x+(D_MAX-1)*WIDTH;
            short cost = cost_all[pos];
            last_line_disparity_last[0] = cost;
            dp_array[pos] = cost;
            last_best_disparity_pos = 0;
            last_best_disparity_value = cost;
            //d > 0
            #pragma unroll
            for(int d=1;d<D_MAX;d++){
                //当前代价
                short cost = cost_all[pos];
                last_line_disparity_last[d] = cost;
                dp_array[pos+FRAME_SIZE*d] = cost;
                //选取最好的保存
                if(cost < last_best_disparity_value){
                    last_best_disparity_pos = d;
                    last_best_disparity_value = cost;
                }
            }
        }
        //y > 0
        #pragma unroll
        for(int y=HEIGHT-2;y>=0;y--){
            x++;
            if(x>=WIDTH){
                return;
            }
            const int pos = y * WIDTH + x;
            //d = 0
            {
                short cost = cost_all[pos];
                //no left disparity
                ///int left_t = P1+last_line_disparity[d-1];
                //mid
                short mid_t = last_line_disparity_last[0];
                //right
                short right_t = last_line_disparity_last[0+1]+P1;
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
                short d0_value = cost+min_t-last_best_disparity_value;
                dp_array[pos] = d0_value;
                now_best_disparity_pos = 0;
                now_best_disparity_value = d0_value;
                last_line_disparity_now[0] = d0_value;
            }
            #pragma unroll
            for(int d=1;d<D_MAX-1;d++){
                int pos_d = pos+FRAME_SIZE*d;
                short cost = cost_all[pos_d];
                short left_t = last_line_disparity_last[d-1]+P1;
                short mid_t = last_line_disparity_last[d];
                short right_t = last_line_disparity_last[d+1]+P1;
                short last_t = last_best_disparity_value+P2;
                //get min of t
                int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
                short di_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = di_value;
                last_line_disparity_now[d] = di_value;
                if(di_value < now_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = di_value;
                }
            }
            //d = D_MAX-1
            {
                int d = D_MAX-1;
                int pos_d = pos+FRAME_SIZE*(D_MAX-1);
                short cost = cost_all[pos_d];
                //no left disparity
                short left_t = last_line_disparity_last[d-1]+P1;
                //mid
                short mid_t = last_line_disparity_last[d];
                //last min
                short last_t = last_best_disparity_value+P2;
                //get min of t
                short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
                short de_value = cost+min_t-last_best_disparity_value;
                dp_array[pos_d] = de_value;
                last_line_disparity_now[d] = de_value;
                if(de_value < last_best_disparity_value){
                    now_best_disparity_pos = d;
                    now_best_disparity_value = de_value;
                }
            }
            //swap
            {
                short *p_t = last_line_disparity_now;
                last_line_disparity_now = last_line_disparity_last;
                last_line_disparity_last = p_t;
                last_best_disparity_pos = now_best_disparity_pos;
                last_best_disparity_value = now_best_disparity_value;
            }
        }
    }
}
__global__ void cost_sum_kernel(int width, int height,const uchar* d_left_texture,uchar* cost_array,uchar* cost_census_array,
                                uchar* cost_all)
{
	
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    int pos = y * width + x;
	if(x < width && y < height) {
	    float texture = d_left_texture[pos];
	    float P1 = texture >= 45?1:texture/45.0;
	    float P2 = 1-P1;
		for(int d=0;d<D_MAX;d++){
		    int pos_d = pos+d*FRAME_SIZE;
		    float cost_census = cost_census_array[pos_d]<<3;
		    float cost_block = cost_array[pos_d];
		    float sum_cost = P1*cost_census+P2*cost_block;
		    cost_all[pos_d] = sum_cost;
		}
	}
}

__global__ void wta_kernel(uchar* dst, int width, int height,short* dp_array,uchar* left_texture)
{
	
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    int pos = y * width + x;
	if(x < width && y < height) {
		int real_disparity = 0,min_cost = 
		    dp_array[pos]+dp_array[pos+D_MAX*FRAME_SIZE]+
		    dp_array[pos+2*D_MAX*FRAME_SIZE]+dp_array[pos+3*D_MAX*FRAME_SIZE]+
		    dp_array[pos+4*D_MAX*FRAME_SIZE]+dp_array[pos+5*D_MAX*FRAME_SIZE];
	    if(left_texture[pos] >0){
		    for(int d=0;d<D_MAX;d++){
		        int pos_now = pos+width*height*d;
		        int vlaue = 
		            dp_array[pos_now]+dp_array[pos_now+D_MAX*FRAME_SIZE]+
		            dp_array[pos_now+2*D_MAX*FRAME_SIZE]+dp_array[pos_now+3*D_MAX*FRAME_SIZE]+
		            dp_array[pos_now+4*D_MAX*FRAME_SIZE]+dp_array[pos_now+5*D_MAX*FRAME_SIZE];
		        //int vlaue = cost_array[pos+width*height*d];
			    if(vlaue < min_cost){
				    min_cost = vlaue;
				    real_disparity = d;
			    }
		    }
		}
		dst[pos] = real_disparity;
	}
}

__global__ void transpose(uchar *odata,uchar *idata)  
{  
    __shared__ float block[16][16];  
    const int BLOCK_DIM = 16;
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;  
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;  
    if((xIndex < WIDTH) && (yIndex < HEIGHT))  
    {  
        unsigned int index_in = yIndex * WIDTH + xIndex;  
        block[threadIdx.y][threadIdx.x] = idata[index_in];  
    }  
  
    __syncthreads();  
  
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;  
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;  
    if((xIndex < HEIGHT) && (yIndex < WIDTH))  
    {  
        unsigned int index_out = yIndex * HEIGHT + xIndex;  
        odata[index_out] = block[threadIdx.x][threadIdx.y];  
    }  
}

extern "C" void diff_caller(const uchar* h_left,uchar* h_left_texture,uchar* d_left_texture,
                            const uchar* left,const uchar* right, const uchar* left_box,const uchar* right_box, 
                            short* h_dp_array,short* dp_array,
                            uchar* dst,
                            uchar* cost_array,
                            uchar* cost_census_array,
                            uchar* h_cost_sum_array,uchar* cost_sum_array,
                            uint* left_census,
                            uint* right_census,
                            int width, int height)
{
    census_caller(left,left_census,width,height);
    //hipDeviceSynchronize();
    census_caller(right,right_census,width,height);
    //hipDeviceSynchronize();
    {
		dim3 block(width-D_MAX,1);
		dim3 grid(1,CELL_DIV(height,block.y*HEIGHT_SINGLE));
		diff_kernel<<<grid,block>>>(left_box,right_box,cost_array);
		//hipDeviceSynchronize();
	}
    low_texture_detection(h_left,h_left_texture,7);
    hipDeviceSynchronize();
    {
		dim3 block(width-D_MAX,1);
		dim3 grid(1,CELL_DIV(height,(HEIGHT_SINGLE*block.y)));
		diff_census_kernel<<<grid,block>>>(left_census,right_census,cost_census_array);
		hipDeviceSynchronize();
	}
	
	/*{
		dim3 block(D_MAX,16);
		dim3 grid(1,CELL_DIV(height,block.y));
		sum_column_kernel<<<grid,block>>>(cost_array);
		hipDeviceSynchronize();
	}
	{
		dim3 block(1,WIDTH);
		dim3 grid(D_MAX,1);
		sum_row_kernel<<<grid,block>>>(cost_array);
		hipDeviceSynchronize();
	}*/
	{
		dim3 block(32,16);
		dim3 grid(CELL_DIV(width-D_MAX,block.x),CELL_DIV(height,block.y));
	    cost_sum_kernel<<<grid,block>>>(width,height,d_left_texture,cost_array,cost_census_array,cost_sum_array);
	    hipDeviceSynchronize();
    }
    //dp_l2r(h_dp_array,h_left_texture,h_cost_sum_array,25,100);
	{
	    dim3 block(WIDTH,1);
		dim3 grid(CELL_DIV(width-D_MAX,block.x),1);
		dp_1_kernel<<<grid,block>>>(dp_array,cost_sum_array);
        dp_2_kernel<<<grid,block>>>(dp_array+D_MAX*FRAME_SIZE,cost_sum_array);
        dp_3_kernel<<<grid,block>>>(dp_array+2*D_MAX*FRAME_SIZE,cost_sum_array);
        dp_4_kernel<<<grid,block>>>(dp_array+3*D_MAX*FRAME_SIZE,cost_sum_array);
        dp_5_kernel<<<grid,block>>>(dp_array+4*D_MAX*FRAME_SIZE,cost_sum_array);
        dp_6_kernel<<<grid,block>>>(dp_array+5*D_MAX*FRAME_SIZE,cost_sum_array);
		hipDeviceSynchronize();
	}
	{
		dim3 block(32,16);
		dim3 grid(CELL_DIV(width-D_MAX,block.x),CELL_DIV(height,block.y));
		wta_kernel<<<grid,block>>>(dst,width,height,dp_array,d_left_texture);

		hipDeviceSynchronize();
	}
}
