#include "hip/hip_runtime.h"
#include "internal.hpp"
using namespace cv;
using namespace cv::cuda;
//the threads number of one block
const int WINDOW_H = 5;
const int WINDOW_W = 7;
const int THREADS_H = 16;
const int THREADS_W = 32;
const int WIDTH_SIZE = THREADS_W + WINDOW_W;
const int HEIGHT_SIZE = THREADS_H + WINDOW_H;
const int WINDOW_H_D2 = WINDOW_H / 2;
const int WINDOW_W_D2 = WINDOW_W / 2;
//窗口 = 5X7 存储空间32bit 中心5点去除，用去30bit 剩余两bit未用
__global__
void census_kernel(const uchar* src, uint* dst, int width, int height){
	const int y = threadIdx.y + blockIdx.y * blockDim.y;
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int pos = x + y * width;
	__shared__ uchar src_s[WIDTH_SIZE*HEIGHT_SIZE];
	if(x < width && y < height)
	{
		//left top side
		{
		    const int lt_y = y - WINDOW_H_D2;
		    const int lt_x = x - WINDOW_W_D2;
		    if (lt_y >= 0 && lt_x >= 0)
		    {
			    src_s[threadIdx.y*WIDTH_SIZE + threadIdx.x] = src[lt_y*width + lt_x];
		    }
	    }
		//right top side
		{
			const int rt_y = y - WINDOW_H_D2;
			const int rt_x = x - WINDOW_W_D2 + blockDim.x;
			if (threadIdx.x + blockDim.x < WIDTH_SIZE && threadIdx.y < HEIGHT_SIZE) {
				if (rt_y >= 0&& rt_x < width) {
					src_s[threadIdx.y*WIDTH_SIZE + threadIdx.x + blockDim.x] = src[rt_y*width + rt_x];
				}
			}
		}
		//left bottom side
		{
		    const int lb_y = y - WINDOW_H_D2 + blockDim.y;
			const int lb_x = x - WINDOW_W_D2;
			if (threadIdx.x < WIDTH_SIZE && threadIdx.y + blockDim.y < HEIGHT_SIZE) {
				if (lb_y < height && lb_x >= 0) {
					src_s[(threadIdx.y + blockDim.y)*WIDTH_SIZE + threadIdx.x] = src[lb_y*width + lb_x];
				}
			}
		}
	    //right bottom side
	    {
			const int rb_y = y - WINDOW_H_D2 + blockDim.y;
			const int rb_x = x - WINDOW_W_D2 + blockDim.x;
			if (threadIdx.x + blockDim.x < WIDTH_SIZE && threadIdx.y + blockDim.y < HEIGHT_SIZE) {
				if (rb_y < height && rb_x < width) {
					src_s[(threadIdx.y + blockDim.y)*WIDTH_SIZE + threadIdx.x + blockDim.x] = src[rb_y*width + rb_x];
				}
			}
		}
		__syncthreads();
		//计算census
	    if (y >= WINDOW_H_D2 && y < height - WINDOW_H_D2 && x >=WINDOW_W_D2 && x < width - WINDOW_W_D2){
	        const int y_running = threadIdx.y + WINDOW_H_D2;
			const int x_running = threadIdx.x + WINDOW_W_D2;
			const int offset = x_running + y_running * WIDTH_SIZE;
		    //五点平均值
			const int c = ((int)src_s[offset]+src_s[offset-1]+src_s[offset+1]+src_s[offset-WIDTH_SIZE]+src_s[offset+WIDTH_SIZE])/5;
		    uint value=0;
		    //计算中心点上方的census
		    for (int y = -WINDOW_H_D2; y < -1; y++) {
		        const int p_y =  WIDTH_SIZE*(y_running + y);
				for (int x = -WINDOW_W_D2; x <= WINDOW_W_D2; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
			}
			
			//计算中心点上方的census y = -1
			{
		        const int p_y =  WIDTH_SIZE*(y_running -1);
		        //x < 0
				for (int x = -WINDOW_W_D2; x < 0; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
				//x > 0
				for (int x = 1; x <=WINDOW_W_D2; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
			}
			
			//计算中心点census y = 0
			{
		        const int p_y =  WIDTH_SIZE*(y_running);
		        //x < 0
				for (int x = -WINDOW_W_D2; x < -1; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
				//x > 0
				for (int x = 2; x <=WINDOW_W_D2; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
			}
			
			//计算中心点下方的census y = 1
			{
		        const int p_y =  WIDTH_SIZE*(y_running + 1);
		        //x < 0
				for (int x = -WINDOW_W_D2; x < 0; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
				//x > 0
				for (int x = 1; x <=WINDOW_W_D2; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
			}
			//计算中心点下方的census
		    for (int y = 2; y <= WINDOW_H_D2; y++) {
		        const int p_y =  WIDTH_SIZE*(y_running + y);
				for (int x = -WINDOW_W_D2; x <= WINDOW_W_D2; x++) {
					uint result = (c - src_s[p_y + x_running + x]) > 0;
					value <<= 1;
					value += result;
				}
			}
			dst[pos] = value;
	    }
	}

};

__global__ void diff_census_kernel(const uint* left_census,const uint*  right_census,uchar *cost_array)
{
	__shared__ uint right_s[HEIGHT_SINGLE][WIDTH];
	uint value_left[HEIGHT_SINGLE];
    const int y_base = (blockIdx.y*blockDim.y+threadIdx.y)*HEIGHT_SINGLE;
	const int x = (blockIdx.x*blockDim.x+threadIdx.x)+D_MAX;
	if (y_base < HEIGHT){
		#pragma unroll
		for(int y=0;y<HEIGHT_SINGLE;y++){
			const int pos_left = (y_base+y) * WIDTH + x;
			value_left[y] = left_census[pos_left];
			right_s[y][x] = right_census[pos_left];
			if(x < 2 * D_MAX){
				right_s[y][(x-D_MAX)] = right_census[pos_left-D_MAX];
			}
		}
		__syncthreads();
		#pragma unroll
		for(int disparity=0;disparity<D_MAX;disparity++){
		    for(int y=0;y<HEIGHT_SINGLE;y++){			
			    const int pos_left = (y_base+y) * WIDTH + x;
			    const int pos_right = x-disparity;
			    const uint value_left_t = value_left[y];
			    const uint value_right_t = right_s[y][x-disparity];
			    if(x < WIDTH && y_base+y < HEIGHT) {
				    cost_array[pos_left+FRAME_SIZE*disparity] = (__popc(value_left_t^value_right_t));
				}
			}
		}
	}
}


__global__
void diff_census(){
	__popc(0);
};

extern "C" void census_caller(const uchar* src,uint* dst,int width, int height){
    const dim3   block(THREADS_W, THREADS_H);
    const dim3   grid(CELL_DIV(width,THREADS_W),CELL_DIV(height,THREADS_H));
    //printf("width:%d,height:%d\r\n",width,height);
    census_kernel<<<grid,block>>>(src,dst,width,height);
}
