#include "hip/hip_runtime.h"
//测试 单一路径规划
__global__ void dp_kernel(uchar* dst,const uchar* cost_array,const uchar* cost_census_array){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    //y = 0
    {
        //d = 0
        short cost = ((cost_census_array[x]<<2)+(cost_array[x]>>1));
        last_line_disparity_last[0] = cost;
        sum_array[x] = cost;
        last_best_disparity_pos = 0;
        last_best_disparity_value = cost;
        //d > 0
        for(int d=1;d<D_MAX;d++){
            //当前代价
            short cost = ((cost_census_array[x+d*FRAME_SIZE]<<2)+(cost_array[x+d*FRAME_SIZE]>>1));
            last_line_disparity_last[d] = cost;
            sum_array[x+FRAME_SIZE*d] = cost;
            //选取最好的保存
            if(cost < last_best_disparity_value){
                last_best_disparity_pos = d;
                last_best_disparity_value = cost;
            }
        }
    }
    //y > 0
    for(int y=1;y<HEIGHT;y++){
        const int pos = y * WIDTH + x;
        //d = 0
        {
            short cost = ((cost_census_array[pos]<<2)+(cost_array[pos]>>1));
            //no left disparity
            ///int left_t = P1+last_line_disparity[d-1];
            //mid
            short mid_t = last_line_disparity_last[0];
            //right
            short right_t = last_line_disparity_last[0+1]+P1;
            //last min
            short last_t = last_best_disparity_value+P2;
            //get min of t
            short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
            short d0_value = cost+min_t-last_best_disparity_value;
            sum_array[pos] = d0_value;
            now_best_disparity_pos = 0;
            now_best_disparity_value = d0_value;
            last_line_disparity_now[0] = d0_value;
        }
        for(int d=1;d<D_MAX-1;d++){
            int pos_d = pos+FRAME_SIZE*d;
            short cost = ((cost_census_array[pos_d]<<2)+(cost_array[pos_d]>>1));
            short left_t = last_line_disparity_last[d-1]+P1;
            short mid_t = last_line_disparity_last[d];
            short right_t = last_line_disparity_last[d+1]+P1;
            short last_t = last_best_disparity_value+P2;
            //get min of t
            int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
            short di_value = cost+min_t-last_best_disparity_value;
            sum_array[pos_d] = di_value;
            last_line_disparity_now[d] = di_value;
            if(di_value < now_best_disparity_value){
                now_best_disparity_pos = d;
                now_best_disparity_value = di_value;
            }
            //dst[pos] = last_best_disparity_pos;
        }
        //d = D_MAX-1
        {
            int d = D_MAX-1;
            int pos_d = pos+FRAME_SIZE*(D_MAX-1);
            short cost = ((cost_census_array[pos_d]<<2)+(cost_array[pos_d]>>1));
            //no left disparity
            short left_t = last_line_disparity_last[d-1]+P1;
            //mid
            short mid_t = last_line_disparity_last[d];
            //last min
            short last_t = last_best_disparity_value+P2;
            //get min of t
            short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
            short de_value = cost+min_t-last_best_disparity_value;
            sum_array[pos_d] = de_value;
            last_line_disparity_now[d] = de_value;
            if(de_value < last_best_disparity_value){
                now_best_disparity_pos = d;
                now_best_disparity_value = de_value;
            }
        }
        //swap
        {
            short *p_t = last_line_disparity_now;
            last_line_disparity_now = last_line_disparity_last;
            last_line_disparity_last = p_t;
            last_best_disparity_pos = now_best_disparity_pos;
            last_best_disparity_value = now_best_disparity_value;
        }
        //dst[pos] = last_best_disparity_pos;
    }
    
    //y = HEIGHT-1
    {
        const int pos = (HEIGHT-1)*WIDTH+x;
        //d = 0
        short cost = ((cost_census_array[pos]<<2)+(cost_array[pos]>>1));
        last_line_disparity_last[0] = cost;
        sum_array[pos] = cost;
        last_best_disparity_pos = 0;
        last_best_disparity_value = cost + sum_array[pos];
        //d > 0
        for(int d=1;d<D_MAX;d++){
            //当前代价
            short cost = ((cost_census_array[pos+d*FRAME_SIZE]<<2)+(cost_array[pos+d*FRAME_SIZE]>>1));
            //calcute after up and down 
            short sum_value = cost + sum_array[pos+FRAME_SIZE*d];
            last_line_disparity_last[d] = cost;
            sum_array[pos+FRAME_SIZE*d] = cost;
            //选取最好的保存
            if(sum_value < last_best_disparity_value){
                last_best_disparity_pos = d;
                last_best_disparity_value = sum_value;
            }
        }
        dst[pos] = last_best_disparity_pos;
    }
    //y > 0
    for(int y=HEIGHT-2;y>=0;y--){
        const int pos = y * WIDTH + x;
        //d = 0
        {
            short cost = ((cost_census_array[pos]<<2)+(cost_array[pos]>>1));
            //no left disparity
            ///int left_t = P1+last_line_disparity[d-1];
            //mid
            short mid_t = last_line_disparity_last[0];
            //right
            short right_t = last_line_disparity_last[0+1]+P1;
            //last min
            short last_t = last_best_disparity_value+P2;
            //get min of t
            short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
            short d0_value = cost+min_t-last_best_disparity_value;
            //calcute after up and down 
            short sum_d0_value = d0_value + sum_array[pos];
            sum_array[pos] = d0_value;
            now_best_disparity_pos = 0;
            now_best_disparity_value = sum_d0_value;
            last_line_disparity_now[0] = d0_value;
        }
        for(int d=1;d<D_MAX-1;d++){
            int pos_d = pos+FRAME_SIZE*d;
            short cost = ((cost_census_array[pos_d]<<2)+(cost_array[pos_d]>>1));
            short left_t = last_line_disparity_last[d-1]+P1;
            short mid_t = last_line_disparity_last[d];
            short right_t = last_line_disparity_last[d+1]+P1;
            short last_t = last_best_disparity_value+P2;
            //get min of t
            int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
            short di_value = cost+min_t-last_best_disparity_value;
            //calcute after up and down 
            short sum_di_value = di_value + sum_array[pos_d];
            sum_array[pos_d] = di_value;
            last_line_disparity_now[d] = di_value;
            if(sum_di_value < now_best_disparity_value){
                now_best_disparity_pos = d;
                now_best_disparity_value = sum_di_value;
            }
        }
        //d = D_MAX-1
        {
            int d = D_MAX-1;
            int pos_d = pos+FRAME_SIZE*(D_MAX-1);
            short cost = ((cost_census_array[pos_d]<<2)+(cost_array[pos_d]>>1));
            //no left disparity
            short left_t = last_line_disparity_last[d-1]+P1;
            //mid
            short mid_t = last_line_disparity_last[d];
            //last min
            short last_t = last_best_disparity_value+P2;
            //get min of t
            short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
            short de_value = cost+min_t-last_best_disparity_value;
            //calcute after up and down 
            short sum_de_value = de_value + sum_array[pos_d];
            sum_array[pos_d] = de_value;
            last_line_disparity_now[d] = de_value;
            if(sum_de_value < last_best_disparity_value){
                now_best_disparity_pos = d;
                now_best_disparity_value = sum_de_value;
            }
        }
        //swap
        {
            short *p_t = last_line_disparity_now;
            last_line_disparity_now = last_line_disparity_last;
            last_line_disparity_last = p_t;
            last_best_disparity_pos = now_best_disparity_pos;
            last_best_disparity_value = now_best_disparity_value;
        }
        dst[pos] = last_best_disparity_pos;
    }
}

//测试 单一路径规划
__global__ void dp_kernel(uchar* dst,const uchar* cost_array,const uchar* cost_census_array){
    int d = blockIdx.x*blockDim.x+threadIdx.x;
    int x = blockIdx.y*blockDim.y+threadIdx.y+D_MAX;
    __shared__ int disparity_s[2][D_MAX];
    int *last_line_disparity_now = disparity_s[0];
    int *last_line_disparity_last = disparity_s[1];
    int left_t,mid_t,right_t;
    int pos = x+d*FRAME_SIZE;
    //y = 0
    {
        //d = 0
        short cost = ((cost_census_array[pos]<<2)+(cost_array[pos]>>1));
        last_line_disparity_last[d] = cost;
        sum_array[pos] = cost;
    }
    __syncthreads();
    //y > 0
    for(int y=1;y<HEIGHT;y++){
        pos = x+d*FRAME_SIZE+y*WIDTH;
        left_t = 0x0ffffff;
        right_t = 0x0ffffff;
        int cost = ((cost_census_array[pos]<<2)+(cost_array[pos]>>1));
        mid_t = last_line_disparity_last[d];
        if(d > 0){
            left_t = last_line_disparity_last[d-1]+P1;
        }
        if(d < D_MAX-1){
            right_t = last_line_disparity_last[d+1]+P1;
        }
        
        //get min of t
        int min_t = min(mid_t,min(left_t,right_t));
        int di_value = cost+min_t;
        sum_array[pos] = di_value;
        last_line_disparity_last[d] = di_value;
        //swap
        {
            int *p_t = last_line_disparity_now;
            last_line_disparity_now = last_line_disparity_last;
            last_line_disparity_last = p_t;
        }
    }
}


//测试 单一路径规划
__global__ void dp_kernel(uchar* dst,const uchar* cost_array,const uchar* cost_census_array){
    int x = blockIdx.x*blockDim.x+threadIdx.x+D_MAX;
    //__shared__ int disparity_s[6*2*WIDTH*D_MAX];
    short last_line_disparity[2][D_MAX];
    short *last_line_disparity_now = last_line_disparity[0];
    short *last_line_disparity_last = last_line_disparity[1];
    
    
    //上次最佳代价位置
    short last_best_disparity_pos = 0;
    //上次最佳代价值
    short last_best_disparity_value = 0;
    //最佳代价位置
    short now_best_disparity_pos = 0;
    //最佳代价值
    short now_best_disparity_value = 0;
    //y = 0
    {
        //d = 0
        short cost = ((cost_census_array[x]<<2)+(cost_array[x]>>1));
        last_line_disparity_last[0] = cost;
        sum_array[x] = cost;
        last_best_disparity_pos = 0;
        last_best_disparity_value = cost;
        //d > 0
        for(int d=1;d<D_MAX;d++){
            //当前代价
            short cost = ((cost_census_array[x+d*FRAME_SIZE]<<2)+(cost_array[x+d*FRAME_SIZE]>>1));
            last_line_disparity_last[d] = cost;
            sum_array[x+FRAME_SIZE*d] = cost;
            //选取最好的保存
            if(cost < last_best_disparity_value){
                last_best_disparity_pos = d;
                last_best_disparity_value = cost;
            }
        }
    }
    //y > 0
    for(int y=1;y<HEIGHT;y++){
        const int pos = y * WIDTH + x;
        //d = 0
        {
            short cost = ((cost_census_array[pos]<<2)+(cost_array[pos]>>1));
            //no left disparity
            ///int left_t = P1+last_line_disparity[d-1];
            //mid
            short mid_t = last_line_disparity_last[0];
            //right
            short right_t = last_line_disparity_last[0+1]+P1;
            //last min
            short last_t = last_best_disparity_value+P2;
            //get min of t
            short min_t = mycuda::min(mid_t,mycuda::min(right_t,last_t));
            short d0_value = cost+min_t-last_best_disparity_value;
            sum_array[pos] = d0_value;
            now_best_disparity_pos = 0;
            now_best_disparity_value = d0_value;
            last_line_disparity_now[0] = d0_value;
        }
        for(int d=1;d<D_MAX-1;d++){
            int pos_d = pos+FRAME_SIZE*d;
            short cost = ((cost_census_array[pos_d]<<2)+(cost_array[pos_d]>>1));
            short left_t = last_line_disparity_last[d-1]+P1;
            short mid_t = last_line_disparity_last[d];
            short right_t = last_line_disparity_last[d+1]+P1;
            short last_t = last_best_disparity_value+P2;
            //get min of t
            int min_t = mycuda::min(mid_t,mycuda::min(left_t,mycuda::min(right_t,last_t)));
            short di_value = cost+min_t-last_best_disparity_value;
            sum_array[pos_d] = di_value;
            last_line_disparity_now[d] = di_value;
            if(di_value < now_best_disparity_value){
                now_best_disparity_pos = d;
                now_best_disparity_value = di_value;
            }
            dst[pos] = last_best_disparity_pos;
        }
        //d = D_MAX-1
        {
            int d = D_MAX-1;
            int pos_d = pos+FRAME_SIZE*(D_MAX-1);
            short cost = ((cost_census_array[pos_d]<<2)+(cost_array[pos_d]>>1));
            //no left disparity
            short left_t = last_line_disparity_last[d-1]+P1;
            //mid
            short mid_t = last_line_disparity_last[d];
            //last min
            short last_t = last_best_disparity_value+P2;
            //get min of t
            short min_t = mycuda::min(mid_t,mycuda::min(left_t,last_t));
            short de_value = cost+min_t-last_best_disparity_value;
            sum_array[pos_d] = de_value;
            last_line_disparity_now[d] = de_value;
            if(de_value < last_best_disparity_value){
                now_best_disparity_pos = d;
                now_best_disparity_value = de_value;
            }
        }
        //swap
        {
            short *p_t = last_line_disparity_now;
            last_line_disparity_now = last_line_disparity_last;
            last_line_disparity_last = p_t;
            last_best_disparity_pos = now_best_disparity_pos;
            last_best_disparity_value = now_best_disparity_value;
        }
        dst[pos] = last_best_disparity_pos;
    }
}
